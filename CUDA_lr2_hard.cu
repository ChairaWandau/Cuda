#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>

#define N 10

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
    int id = threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void generate(hiprandState* globalState, float* randomArray)
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform(&localState);
    randomArray[ind] = RANDOM;
    globalState[ind] = localState;
}

int main(int argc, char** argv)
{
    dim3 T(N, 1, 1);
    hiprandState* devStates;
    //массив рандомных чисел
    float* randomValues = new float[N];
    //копия массива рандомных чисел
    float* devRandomValues;
    //выделяем память
    hipMalloc(&devStates, N * sizeof(hiprandState));
    hipMalloc(&devRandomValues, N * sizeof(*randomValues));
    //устанавливаем seeds
    setup_kernel << <1, T >> > (devStates, time(NULL));
    //Генерируем рандомные числа
    generate << <1, T >> > (devStates, devRandomValues);
    //копируем значения
    hipMemcpy(randomValues, devRandomValues, N * sizeof(*randomValues), hipMemcpyDeviceToHost);
    //выводим на экран
    for (int i = 0; i < N; i++)
    {
        printf("%f\n", randomValues[i]);
    }
    //освобождаем память
    hipFree(devRandomValues);
    hipFree(devStates);
    delete randomValues;
    getchar();
    return 0;
}