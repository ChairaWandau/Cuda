#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
__global__ void Dispersion(float *a, int* m, float* dispersion) {
	// Мат ожидание
	float MX = 0;
	// Мат ожидание от X^2
	float MX_2 = 0;
	// Перебираем массив, находя мат ожидания
	for (int j = 0; j < *m; j++)
	{
		MX += a[0 * *m + j] * a[1 * *m + j];
		MX_2 += a[0 * *m + j]* a[0 * *m + j] * a[1 * *m + j];
	}
	//Считаем дисперсию
	*dispersion = MX_2 - MX * MX;
}
int main(void) {
	int i, j; // указатели на элемент массива
	int n=2, m; // количество строк и столбцов
	int* dev_m; // device копия m
	float dispersion;  // результат расчета дисперсии
	float* dev_dispersion;  // device копия результата расчета дисперсии
	printf("Enter number of columns: ");
	scanf("%d", &m);
	float* a = new float[n * m];  // указатель на массив
	float* dev_a = new float[n * m]; // device копия a
	// Ввод элементов массива
	for (i = 0; i < n; i++)  // цикл по строкам
	{
		for (j = 0; j < m; j++)  // цикл по столбцам
		{
			printf("a[%d][%d] = ", i, j);
			scanf("%f", (float*)(a + i * m + j));
		}
	}
	//выделяем память
	hipMalloc((void**)&dev_a, n*m*sizeof(float));
	hipMalloc((void**)&dev_m, sizeof(int));
	hipMalloc((void**)&dev_dispersion, sizeof(float));
	// копируем ввод на device
	hipMemcpy(dev_a, a, n * m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_m, &m, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_dispersion, &dispersion, sizeof(float), hipMemcpyHostToDevice);
	// запускаем Dispersion() на GPU, передавая параметры
	Dispersion << < 1, 1 >> > (dev_a, dev_m, dev_dispersion);
	// копируем результат функции обратно
	hipMemcpy(&dispersion, dev_dispersion, sizeof(float), hipMemcpyDeviceToHost);
	// освобождаем память
	hipFree(dev_a);
	hipFree(dev_m);
	hipFree(dev_dispersion);
	// выводим результат на экран
	printf("D = %f\n", dispersion);
	return 0;
}